
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

__global__ void get_res(int *locks, int *keys, int keyc, int *result) {
    int id = blockIdx.x * blockDim.x + threadIdx.x, out = 0, i;
    int lock = locks[id];
    for (i = 0; i < keyc; ++i) out += !((lock + keys[i]) & 559240);
    atomicAdd(result, out);
}

int main() {
    FILE *f = fopen("in.txt", "r");
    char buf[44];
    int lockc = 0, keyc = 0, locks[10000], keys[10000], *l_cu, *k_cu, *result, i, j, k;
    while (fread(buf, sizeof(char), 43, f)) {
        for (i = k = 0; i < 5; ++i) {
            j = (buf[i] == '#') + (buf[i + 6] == '#') + (buf[i + 12] == '#') + (buf[i + 18] == '#') + (buf[i + 24] == '#') + (buf[i + 30] == '#') + (buf[i + 36] == '#');
            k += j << (4 * i);
        }
        j = ((i = buf[0] == '#') ? lockc : keyc)++;
        (i ? locks : keys)[j] = k;
    }
    hipMalloc(&l_cu, sizeof(int) * lockc);
    hipMalloc(&k_cu, sizeof(int) * keyc);
    hipMallocManaged(&result, sizeof(int));
    hipMemcpy(l_cu, locks, sizeof(int) * lockc, hipMemcpyHostToDevice);
    hipMemcpy(k_cu, keys, sizeof(int) * keyc, hipMemcpyHostToDevice);
    get_res<<<1, lockc>>>(l_cu, k_cu, keyc, result);
    hipDeviceSynchronize();
    hipFree(l_cu);
    hipFree(k_cu);
    printf("%d\n", *result);
    hipFree(result);
}